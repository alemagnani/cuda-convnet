
#include <sparse_nvmatrix.cuh>
#include <hipsparse.h>
#include "nvmatrix.cuh"
#include "cuda_setup.cuh"

SparseNVMatrix::SparseNVMatrix() {
	_sparseInd = NULL;
	_sparsePtr = NULL;

	_ownsDataInd = true;
	_ownsDataPtr = true;
	_nzz = 0;
	_sparse_type = SparseMatrix::CSC;

}

SparseNVMatrix::~SparseNVMatrix() {
	//cout << "sparse delete\n";
	if (_ownsDataInd && _numElements > 0) {
		hipblasStatus_t status = cublasFree(_sparseInd);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! memory free error on delete _sparseInd\n");
			exit(EXIT_FAILURE);
		}
	}
	if (_ownsDataPtr && _numElements > 0) {
		hipblasStatus_t status = cublasFree(_sparsePtr);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! memory free error on delete _sparsePtr\n");
			exit(EXIT_FAILURE);
		}
	}

}

SparseNVMatrix::SparseNVMatrix(float* devData,int* sparseInd, int* sparsePtr,  int numRows, int numCols, int nzz, SparseMatrix::SPARSE_TYPE type) : NVMatrix( devData, 1, nzz,  1, false) {
	//cout << "new sparse matrix of size " << numRows << " , " << numCols << "\n";
	_nzz = nzz;
	_numRows =  numRows;
	_numCols = numCols;
	_numElements = _numRows * _numCols;
	_ownsDataInd = false;
	_ownsDataPtr = false;
	_sparseInd = sparseInd;
	_sparsePtr = sparsePtr;
	_isTrans = false;
	_sparse_type = type;
}

void SparseNVMatrix::copyFromHost(const SparseMatrix& hostMatrix) {
	assert(hostMatrix.get_non_zeros() == _nzz);
	assert(isSameDims(hostMatrix));
	assert(hostMatrix.get_sparse_type() == get_sparse_type());

	if (_nzz > 0) {
		checkCudaErrors(hipMemcpy(_devData, hostMatrix.getData(),
				sizeof(float) * _nzz , hipMemcpyHostToDevice));

		checkCudaErrors(hipMemcpy(_sparseInd, hostMatrix.getSparseInd(),
				sizeof(int) * _nzz , hipMemcpyHostToDevice));

		checkCudaErrors(hipMemcpy( _sparsePtr, hostMatrix.getSparsePtr(),
				sizeof(int) * ( (get_sparse_type() == SparseMatrix::CSC ?getNumCols() : getNumRows())+1) , hipMemcpyHostToDevice));

	}
}


NVMatrix& SparseNVMatrix::sliceCols(int startCol, int endCol) const{
	//cout << "slicing cols with no target\n\n";
	if (_sparse_type == SparseMatrix::CSR){
		throw string("CSR is not supported for column slicing");
	}

	//cout << "start col "<< startCol << " endcol: " << endCol << " rows: " << getNumRows() << " cols: " << getNumCols() << " nzz: "<< _nzz <<"\n";
	int begin=0;
	int* d_answer;
	checkCudaErrors(hipMalloc(&d_answer, sizeof(int)));

	read_one_entry<<<1,1>>>(_sparsePtr,startCol, d_answer);
	checkCudaErrors(hipMemcpy(&begin, d_answer, sizeof(int), hipMemcpyDeviceToHost));

	int end=0;
	read_one_entry<<<1,1>>>(_sparsePtr,endCol, d_answer);
	checkCudaErrors(hipMemcpy(&end, d_answer, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_answer));
	//hipDeviceSynchronize();
	//cout << "begin" << begin <<"\n";
	//cout << "end" << end <<"\n";
	const int nzz = end -begin;
	//cout << "the slice of sparse as nzz " <<  nzz <<"\n";
	return * new SparseNVMatrix(_devData,_sparseInd , _sparsePtr+ startCol,  getNumRows(), (endCol-startCol), nzz, SparseMatrix::CSC);
}


void SparseNVMatrix::copyFromHost(const Matrix& hostMatrix) {
	copyFromHost((SparseMatrix&) hostMatrix);
}
void SparseNVMatrix::copyFromHost(const Matrix& hostMatrix,
		bool resizeDeviceMatrix) {
	if (resizeDeviceMatrix) {
		resize((SparseMatrix&) hostMatrix);
	}
	copyFromHost(hostMatrix);
}


bool SparseNVMatrix::resize(const SparseMatrix &like) {
	//cout << "resize data of sparse matrix\n";
	bool reallocated = false;
	if (like.get_non_zeros() != _nzz) {
		assert(_ownsData);
		assert(_ownsDataInd);
		assert(_ownsDataPtr);

		_numRows = like.getNumRows();
		_numCols = like.getNumCols();

		if (_nzz > 0) { // free old memory
			//cout << "clearing memory from old psarse matrix during resize\n";
			hipblasStatus_t status = cublasFree(_devData);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! memory free error during resize: %X\n", status);
				exit(EXIT_FAILURE);
			}
			status = cublasFree(_sparseInd);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! memory free error during resize: %X\n", status);
				exit(EXIT_FAILURE);
			}
			status = cublasFree(_sparsePtr);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! memory free error during resize: %X\n", status);
				exit(EXIT_FAILURE);
			}

		}
		_nzz = like.get_non_zeros();
		if (_nzz > 0) { // allocate new memory
			printf("allocating new memory\n");
			hipblasStatus_t status = cublasAlloc(_nzz, sizeof(float),
					(void**) &_devData);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! device memory allocation error\n");
				exit(EXIT_FAILURE);
			}
			status = cublasAlloc(_nzz, sizeof(int), (void**) &_sparseInd);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! device memory allocation error\n");
				exit(EXIT_FAILURE);
			}
			const int s =   (get_sparse_type() == SparseMatrix::CSC ? like.getNumCols() : like.getNumRows()) + 1;
			status = cublasAlloc(s , sizeof(int),
					(void**) &_sparsePtr);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! device memory allocation error\n");
				exit(EXIT_FAILURE);
			}

		} else {
			_devData = NULL;
			_sparseInd = NULL;
			_sparsePtr = NULL;
			_nzz = 0;
		}
		reallocated = true;

		_numRows =  like.getNumRows();
		_numCols = like.getNumCols();
		_numElements = _numRows * _numCols;
		_sparse_type = like.get_sparse_type();
		_isTrans = false;

	}
	return true;
}


/*
 * Does SOFT transpose and returns result, leaving this matrix unchanged
 */
NVMatrix& SparseNVMatrix::getTranspose(){
	//cout << "get transpose of sparse " << get_sparse_type() << "\n";
	return * new SparseNVMatrix(_devData,_sparseInd, _sparsePtr,   getNumCols(), getNumRows(), _nzz, (get_sparse_type() == SparseMatrix::CSR) ? SparseMatrix::CSC : SparseMatrix::CSR );
}

/*
 * Does HARD transpose and puts result in target
 */
void SparseNVMatrix::transpose(NVMatrix& target){
	//cout << "sparse not implemented transpose with target args\n";
	throw string("Not implemented!");
}
/*
 * Does SOFT transpose
 */
void SparseNVMatrix::transpose(){
	if (_sparse_type == SparseMatrix::CSC){
		_sparse_type = SparseMatrix::CSR;
	}else{
		_sparse_type = SparseMatrix::CSC;
	}
	int numColsTmp = getNumCols();
	_numCols = getNumRows();
	_numRows = numColsTmp;
}

bool SparseNVMatrix::transpose(bool trans){
	bool oldTrans = get_sparse_type() == SparseMatrix::CSR;
	    if (oldTrans != trans) {
	    	//cout << "doing transpose for sparse with bool argument\n";
	        transpose();
	    }
	    return oldTrans;
}
void SparseNVMatrix::rightMult(const NVMatrix &b, float scaleAB, NVMatrix &target) const{
	//cout << "right multiplication for sparse " << get_sparse_type() << "\n";
	addProductChanged(b, 0, scaleAB, target);
}


void SparseNVMatrix::addProductChanged( const NVMatrix &b, float scaleTarget, float scaleAB, NVMatrix &target)const{
	//cout << "addproduct changes Sparse\n";
	assert(_numCols == b.getNumRows());
	if(scaleTarget == 0.0) {
		//cout << "target is zero\n";
		target.resize(_numRows, b.getNumCols());
		target.setTrans(true);
	}else{
		assert(target.isTrans());
	}
	assert(target.getNumRows() == _numRows);
	assert(target.getNumCols() == b.getNumCols());
	assert(_numCols == b.getNumRows());


	if (_sparse_type == SparseMatrix::CSC){
		//cout << "csc\n";
		hipsparseStatus_t cusparseStatus = hipsparseScsrmm2(cudaSetup::_cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE ,b.isTrans()?HIPSPARSE_OPERATION_NON_TRANSPOSE: HIPSPARSE_OPERATION_TRANSPOSE,
				getNumCols(), b.getNumCols(), getNumRows(),_nzz,
				&scaleAB, cudaSetup::_sparseDescr,
				getDevData(), _sparsePtr, _sparseInd,
				b.getDevData(),  b.getLeadingDim() ,
				&scaleTarget,
				target.getDevData(), getNumRows());
		checkCudaErrors(cusparseStatus);
	}else{
		//cout << "csr, rows " << getNumRows()<< ", cols: "<< getNumCols() << ", nzz: "<< _nzz <<"\n";
		hipsparseStatus_t cusparseStatus = hipsparseScsrmm2(cudaSetup::_cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE ,b.isTrans()?HIPSPARSE_OPERATION_NON_TRANSPOSE: HIPSPARSE_OPERATION_TRANSPOSE,
				getNumRows(), b.getNumCols(), getNumCols(),_nzz,
				&scaleAB, cudaSetup::_sparseDescr,
				getDevData(), _sparsePtr, _sparseInd,
				b.getDevData(),  b.getLeadingDim() ,
				&scaleTarget,
				target.getDevData(), getNumRows());
		checkCudaErrors(cusparseStatus);
	}

	//cout << "done addproduct changes Sparse\n";

}

void SparseNVMatrix::rightMult(const NVMatrix &b, float scaleAB){
	throw string("rightMult Not implemented for sparse!");
}



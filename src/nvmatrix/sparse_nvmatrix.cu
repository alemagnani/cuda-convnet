
#include <sparse_nvmatrix.cuh>
#include <hipsparse.h>
#include "nvmatrix.cuh"
#include "cuda_setup.cuh"

SparseNVMatrix::SparseNVMatrix() {
	_sparseInd = NULL;
	_sparsePtr = NULL;

	_ownsDataInd = true;
	_ownsDataPtr = true;
	_nzz = 0;
	_sparse_type = SparseMatrix::CSC;

}

SparseNVMatrix::~SparseNVMatrix() {
	//cout << "sparse delete\n";
	if (_ownsDataInd && _numElements > 0) {
		cout << "freeing indeces for sparse matrix \n\n";
		hipblasStatus_t status = cublasFree(_sparseInd);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! memory free error on delete _sparseInd\n");
			exit(EXIT_FAILURE);
		}
	}
	if (_ownsDataPtr && _numElements > 0) {
		cout << "freeing pointer for sparse matrix \n\n";
		hipblasStatus_t status = cublasFree(_sparsePtr);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! memory free error on delete _sparsePtr\n");
			exit(EXIT_FAILURE);
		}
	}

}

SparseNVMatrix::SparseNVMatrix(float* devData,int* sparseInd, int* sparsePtr,  int numRows, int numCols, int nzz, SparseMatrix::SPARSE_TYPE type) : NVMatrix( devData, 1, nzz,  1, false) {
	//cout << "new sparse matrix of size " << numRows << " , " << numCols << "\n";
	_nzz = nzz;
	_numRows =  numRows;
	_numCols = numCols;
	_numElements = _numRows * _numCols;
	_ownsDataInd = false;
	_ownsDataPtr = false;
	_sparseInd = sparseInd;
	_sparsePtr = sparsePtr;
	_isTrans = false;
	_sparse_type = type;
}

void SparseNVMatrix::copyFromHost(const SparseMatrix& hostMatrix) {
	assert(hostMatrix.get_non_zeros() == _nzz);
	assert(isSameDims(hostMatrix));
	assert(hostMatrix.get_sparse_type() == get_sparse_type());

	if (_nzz > 0) {
		checkCudaErrors(hipMemcpy(_devData, hostMatrix.getData(),
				sizeof(float) * _nzz , hipMemcpyHostToDevice));

		checkCudaErrors(hipMemcpy(_sparseInd, hostMatrix.getSparseInd(),
				sizeof(int) * _nzz , hipMemcpyHostToDevice));

		checkCudaErrors(hipMemcpy( _sparsePtr, hostMatrix.getSparsePtr(),
				sizeof(int) * ( (get_sparse_type() == SparseMatrix::CSC ?getNumCols() : getNumRows())+1) , hipMemcpyHostToDevice));

	}
}


NVMatrix& SparseNVMatrix::sliceCols(int startCol, int endCol) const{
	//cout << "slicing cols with no target\n\n";
	if (_sparse_type == SparseMatrix::CSR){
		throw string("CSR is not supported for column slicing");
	}

	//cout << "start col "<< startCol << " endcol: " << endCol << " rows: " << getNumRows() << " cols: " << getNumCols() << " nzz: "<< _nzz <<"\n";
	int begin=0;
	int* d_answer;
	checkCudaErrors(hipMalloc(&d_answer, sizeof(int)));

	read_one_entry<<<1,1>>>(_sparsePtr,startCol, d_answer);
	checkCudaErrors(hipMemcpy(&begin, d_answer, sizeof(int), hipMemcpyDeviceToHost));

	int end=0;
	read_one_entry<<<1,1>>>(_sparsePtr,endCol, d_answer);
	checkCudaErrors(hipMemcpy(&end, d_answer, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_answer));
	//hipDeviceSynchronize();
	//cout << "begin" << begin <<"\n";
	//cout << "end" << end <<"\n";
	const int nzz = end -begin;
	//cout << "the slice of sparse as nzz " <<  nzz << " the old nzz was "<< _nzz << " start col "<< startCol <<"\n";
	return * new SparseNVMatrix(_devData,_sparseInd , _sparsePtr+ startCol,  getNumRows(), (endCol-startCol), nzz, SparseMatrix::CSC);
}


void SparseNVMatrix::copyFromHost(const Matrix& hostMatrix) {
	copyFromHost((SparseMatrix&) hostMatrix);
}
void SparseNVMatrix::copyFromHost(const Matrix& hostMatrix,
		bool resizeDeviceMatrix) {
	if (resizeDeviceMatrix) {
		resize((SparseMatrix&) hostMatrix);
	}
	copyFromHost(hostMatrix);
}


bool SparseNVMatrix::resize(const SparseMatrix &like) {
	//cout << "resize data of sparse matrix\n";
	bool reallocated = false;
	if (like.get_non_zeros() != _nzz) {
		assert(_ownsData);
		assert(_ownsDataInd);
		assert(_ownsDataPtr);

		_numRows = like.getNumRows();
		_numCols = like.getNumCols();

		if (_nzz > 0) { // free old memory
			//cout << "clearing memory from old psarse matrix during resize\n";
			hipblasStatus_t status = cublasFree(_devData);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! memory free error during resize: %X\n", status);
				exit(EXIT_FAILURE);
			}
			status = cublasFree(_sparseInd);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! memory free error during resize: %X\n", status);
				exit(EXIT_FAILURE);
			}
			status = cublasFree(_sparsePtr);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! memory free error during resize: %X\n", status);
				exit(EXIT_FAILURE);
			}

		}
		_nzz = like.get_non_zeros();
		if (_nzz > 0) { // allocate new memory
			//printf("allocating new memory\n");
			hipblasStatus_t status = cublasAlloc(_nzz, sizeof(float),
					(void**) &_devData);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! device memory allocation error\n");
				exit(EXIT_FAILURE);
			}
			status = cublasAlloc(_nzz, sizeof(int), (void**) &_sparseInd);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! device memory allocation error\n");
				exit(EXIT_FAILURE);
			}
			const int s =   (like.get_sparse_type() == SparseMatrix::CSC ? like.getNumCols() : like.getNumRows()) + 1;
			//cout << "allocating for matrix in device " << s << "cols\n";
			status = cublasAlloc(s , sizeof(int),
					(void**) &_sparsePtr);
			if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr, "!!!! device memory allocation error\n");
				exit(EXIT_FAILURE);
			}

		} else {
			_devData = NULL;
			_sparseInd = NULL;
			_sparsePtr = NULL;
			_nzz = 0;
		}
		reallocated = true;

		_numRows =  like.getNumRows();
		_numCols = like.getNumCols();
		_numElements = _numRows * _numCols;
		_sparse_type = like.get_sparse_type();
		_isTrans = false;

	}
	return true;
}


/*
 * Does SOFT transpose and returns result, leaving this matrix unchanged
 */
NVMatrix& SparseNVMatrix::getTranspose(){
	//cout << "get transpose of sparse " << get_sparse_type() << "\n";
	return * new SparseNVMatrix(_devData,_sparseInd, _sparsePtr,   getNumCols(), getNumRows(), _nzz, (get_sparse_type() == SparseMatrix::CSR) ? SparseMatrix::CSC : SparseMatrix::CSR );
}

/*
 * Does HARD transpose and puts result in target
 */
void SparseNVMatrix::transpose(NVMatrix& target){
	//cout << "sparse not implemented transpose with target args\n";
	throw string("Not implemented!");
}
/*
 * Does SOFT transpose
 */
void SparseNVMatrix::transpose(){
	if (_sparse_type == SparseMatrix::CSC){
		_sparse_type = SparseMatrix::CSR;
	}else{
		_sparse_type = SparseMatrix::CSC;
	}
	int numColsTmp = getNumCols();
	_numCols = getNumRows();
	_numRows = numColsTmp;
}

bool SparseNVMatrix::transpose(bool trans){
	bool oldTrans = get_sparse_type() == SparseMatrix::CSR;
	if (oldTrans != trans) {
		//cout << "doing transpose for sparse with bool argument\n";
		transpose();
	}
	return oldTrans;
}
void SparseNVMatrix::rightMult(const NVMatrix &b, float scaleAB, NVMatrix &target) const{
	//cout << "right multiplication for sparse " << get_sparse_type() << "\n";
	addProductChanged(b, 0, scaleAB, target);
}


void SparseNVMatrix::addProductChanged( const NVMatrix &b, float scaleTarget, float scaleAB, NVMatrix &target)const{
	//cout << "addproduct changes Sparse\n";
	assert(_numCols == b.getNumRows());
	if(scaleTarget == 0.0) {
		//cout << "target is zero\n";
		target.resize(_numRows, b.getNumCols());
		target.setTrans(true);
	}else{
		assert(target.isTrans());
	}
	assert(target.getNumRows() == _numRows);
	assert(target.getNumCols() == b.getNumCols());
	assert(_numCols == b.getNumRows());

	//cout << "this:"<< getNumRows()<< ", "<< getNumCols()<< " b: "<<  b.getNumRows()<< ", "<< b.getNumCols() << " target:"<< target.getNumRows()<< ", "<< target.getNumCols()<< " nzz: "<< _nzz<<"\n";
	//cout << "b leading " <<  b.getLeadingDim() <<"\n";

	if (_sparse_type == SparseMatrix::CSC){

		//copyToHost(*(new Matrix()),true);
		//target.copyToHost(*(new Matrix()),true);
		//b.copyToHost(*(new Matrix()),true);
		//hipsparseSetPointerMode(cudaSetup::_cusparseHandle,HIPSPARSE_POINTER_MODE_HOST);
		//juto to check
		/*
		cout << "csc, rows " << getNumRows()<< ", cols: "<< getNumCols() << ", nzz: "<< _nzz<< " btrans: "<< b.isTrans() <<"\n";

		check_matrix<<<1,1>>>(getDevData(), _sparseInd, _sparsePtr, _nzz, getNumCols(), getNumRows());

		check_matrix_dense<<<1,1>>>(b.getDevData(), b.getNumRows(), b.getNumCols());
		check_matrix_dense<<<1,1>>>(target.getDevData(), target.getNumRows(), target.getNumCols());
        */
		if (b.isTrans()){

			//cout << "new implementation col by col";

			sparseMult(cudaSetup::_cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE,
							getNumCols(), b.getNumCols(), getNumRows(),_nzz,
							&scaleAB, cudaSetup::_sparseDescr,
							getDevData(), _sparsePtr, _sparseInd,
							b.getDevData(),  b.getLeadingDim() ,
							&scaleTarget,
							target.getDevData(), getNumRows());


			/*
			hipsparseStatus_t cusparseStatus = hipsparseScsrmm(cudaSetup::_cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE ,
							getNumCols(), b.getNumCols(), getNumRows(),_nzz,
							&scaleAB, cudaSetup::_sparseDescr,
							getDevData(), _sparsePtr, _sparseInd,
							b.getDevData(),  b.getLeadingDim() ,
							&scaleTarget,
							target.getDevData(), getNumRows());
			checkCudaErrors(cusparseStatus);
			*/

		}else{
		hipsparseStatus_t cusparseStatus = hipsparseScsrmm2(cudaSetup::_cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE ,b.isTrans()?HIPSPARSE_OPERATION_NON_TRANSPOSE: HIPSPARSE_OPERATION_TRANSPOSE,
				getNumCols(), b.getNumCols(), getNumRows(),_nzz,
				&scaleAB, cudaSetup::_sparseDescr,
				getDevData(), _sparsePtr, _sparseInd,
				b.getDevData(),  b.getLeadingDim() ,
				&scaleTarget,
				target.getDevData(), getNumRows());
		checkCudaErrors(cusparseStatus);
		}

	}else{
		//cout << "csr, rows " << getNumRows()<< ", cols: "<< getNumCols() << ", nzz: "<< _nzz<< " btrans: "<< b.isTrans() <<"\n";
		//check_matrix<<<1,1>>>(getDevData(), _sparseInd, _sparsePtr, _nzz,  getNumRows(),getNumCols());
		hipsparseStatus_t cusparseStatus = hipsparseScsrmm2(cudaSetup::_cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE ,b.isTrans()?HIPSPARSE_OPERATION_NON_TRANSPOSE: HIPSPARSE_OPERATION_TRANSPOSE,
				getNumRows(), b.getNumCols(), getNumCols(),_nzz,
				&scaleAB, cudaSetup::_sparseDescr,
				getDevData(), _sparsePtr, _sparseInd,
				b.getDevData(),  b.getLeadingDim() ,
				&scaleTarget,
				target.getDevData(), getNumRows());
		checkCudaErrors(cusparseStatus);
	}

	//cout << "done addproduct changes Sparse\n";
}

void SparseNVMatrix::copyToHost(Matrix& hostMatrix, bool resizeTarget) const {
    if (resizeTarget) {
        hostMatrix.resize(_numRows, _numCols);
    }
    copyToHost(hostMatrix);
}


void SparseNVMatrix::copyToHost(Matrix& hostMatrix) const{
	//hostMatrix.apply(NVMatrixOps::Zero());
	NVMatrix tmpStorage = new NVMatrix();
	int r;
	int c;
	if (get_sparse_type() == SparseMatrix::CSC){
		r = getNumCols();
		c = getNumRows();
	}else{
		r = getNumRows();
		c = getNumCols();
	}
	tmpStorage.resize(r, c);
	tmpStorage.setTrans(true);

	cout<< "sparse to dense\n";
	hipsparseStatus_t cusparseStatus = hipsparseScsr2dense( cudaSetup::_cusparseHandle,
			r, c,
			cudaSetup::_sparseDescr,
			getDevData(),
			_sparsePtr, _sparseInd,
			tmpStorage.getDevData(), r);
	checkCudaErrors(cusparseStatus);
	if (get_sparse_type() == SparseMatrix::CSC){
		tmpStorage.transpose();
	}
	tmpStorage.copyToHost(hostMatrix);
}

void SparseNVMatrix::rightMult(const NVMatrix &b, float scaleAB){
	throw string("rightMult Not implemented for sparse!");
}



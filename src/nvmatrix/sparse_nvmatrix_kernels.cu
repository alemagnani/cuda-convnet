#include "hip/hip_runtime.h"


#include "sparse_nvmatrix_kernels.cuh"

__global__  void slice_kernel(int start, int end, float * data, int* ind, int* ptr, float * dest, int stride_dest, bool isTrans){
	const int size = end - start;


	for (int i = blockIdx.x; i < size; i += gridDim.x){
		const int begin = ptr[i+start];
		const int num_entries = ptr[i+1+start] - begin;
		for (int pos = threadIdx.x; pos < num_entries; pos += blockDim.x){
			const int k = pos+begin;
			if (isTrans){
				dest[i + stride_dest * ind[k]] = data[k];
			}else{
				dest[i * stride_dest + ind[k]] = data[k];
			}
		}
	}
}

__global__ void read_one_entry(int* array, int pos_to_read, int * read_value){
	read_value[0] = array[pos_to_read];
}

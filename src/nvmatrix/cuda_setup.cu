#include "cuda_setup.cuh"

#include <iostream>
#include <hipsparse.h>


namespace cudaSetup {


bool            _cudaInitialized = false;
hipsparseHandle_t  _cusparseHandle;
hipsparseMatDescr_t  _sparseDescr;


void CudaStart()
{
	if (!_cudaInitialized) {
		std::cout << "initizlizin cusparse\n\n";
		hipsparseStatus_t cusparseStatus = hipsparseCreate(&_cusparseHandle);
		if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) {
			std::cout << "CUSPARSE initialization failed" << std::endl;
			exit(1);
		}

		_sparseDescr = 0;
		cusparseStatus = hipsparseCreateMatDescr(&_sparseDescr);
		if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) {
			std::cout << "CUSPARSE initialization failed" << std::endl;
			exit(1);
		}
		hipsparseSetMatType(_sparseDescr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(_sparseDescr,HIPSPARSE_INDEX_BASE_ZERO);
		_cudaInitialized = true;
	}
}


void CudaStop()
{
	if (_cudaInitialized)
		hipsparseDestroy(_cusparseHandle);
	_cudaInitialized = false;
}


}



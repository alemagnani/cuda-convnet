#include "hip/hip_runtime.h"


#include "sparse_nvmatrix_kernels.cuh"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include "nvmatrix.cuh"

#include <iostream>

using namespace std;

__global__  void slice_kernel(int start, int end, float * data, int* ind, int* ptr, float * dest, int stride_dest, bool isTrans){
	const int size = end - start;


	for (int i = blockIdx.x; i < size; i += gridDim.x){
		const int begin = ptr[i+start];
		const int num_entries = ptr[i+1+start] - begin;
		for (int pos = threadIdx.x; pos < num_entries; pos += blockDim.x){
			const int k = pos+begin;
			if (isTrans){
				dest[i + stride_dest * ind[k]] = data[k];
			}else{
				dest[i * stride_dest + ind[k]] = data[k];
			}
		}
	}
}

__global__ void check_matrix_dense(float * data, int row, int col){
	printf("checking dense matrix!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!");
	for (int r =0; r < row; r++){
		for(int c=0; c < col; c++){
			float b = data[r + c * row];
			if (isnan(b)){
				printf("foud nana element\n");
				return;
			}
		}
	}
}


__global__ void check_matrix(float * data, int* ind, int* ptr, int nzz, int size, int size2){
	printf("checking sparse matrix!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
	int nzz2 = ptr[size] -ptr[0];
	if (nzz2 != nzz){
		printf("the nzz sizes don't match %d, %d\n",nzz, nzz2);
		return;
	}

	for( int i=0 ; i < size; i++){
		int begin = ptr[i];
		int end = ptr[i+1];

		if (end < begin){
			printf("end before begin\n");
			return;
		}
		if (begin < 0){
			printf("begin less than 0");
			return;
		}


		for (int k= begin; k < end; k++){
			float b = data[k];
			if (isnan(b)){
				printf("foud nan element\n");
				return;
			}
			int pos = ind[k];
			float c = 1.2* b;
			data[k] = c;

			if (pos < 0){
				printf("the position is less than 0");
				return;
			}
			if (pos >= size2){
				printf("pos is bigger or equalt than size2");
				return;
			}

		}


	}

	printf("DONE checking sparse matrix\n");
}

// performs C =  alpha A^T B + C where A is CSR sparse matr and B and C a re dense matrices in column major order A is m X n and  B is m X k and C is n * k

 // this implementation uses shared memory to cache the values of B

__global__  void sparse_mul_trans(float alpha, int m, int n, int k,const float * data,const int* ind,const int* ptr, const float* B, float* C){
	extern __shared__ volatile float b_row_cache[];
	for(int row = blockIdx.x; row < m; row += gridDim.x){
		//read b times alphs into cache
		for (int colB = threadIdx.x; colB < k; colB += blockDim.x){
			b_row_cache[colB] = alpha * B[row + m * colB];
		}
		__syncthreads();
		const int begin = ptr[row];
		const int num_entries = ptr[row+1] - begin;
		for (int pos = threadIdx.x; pos < num_entries; pos += blockDim.x){
			const int kpos = pos+begin;
			const int target_row = ind[kpos];
			const float val = data[kpos];

			for (int colB = 0; colB < k; colB += 1){
				atomicAdd(C+(target_row + colB * n), val * b_row_cache[colB]);
			}
		}
	}
}


/*
__global__  void sparse_mul_trans(float alpha, int m, int n, int k,const float * data,const int* ind,const int* ptr, const float* B, float* C){

	for(int row = blockIdx.x; row < m; row += gridDim.x){
		const int begin = ptr[row];
		const int num_entries = ptr[row+1] - begin;
		for (int pos = threadIdx.x; pos < num_entries; pos += blockDim.x){
			const int kpos = pos + begin;
			const int target_row = ind[kpos];
			const float val = data[kpos] * alpha;

			for (int colB = 0; colB < k; colB += 1){
				atomicAdd(C+(target_row + colB * n), val * B[row + m * colB]);
			}
		}
	}
}
*/


__global__ void read_one_entry(int* array, int pos_to_read, int * read_value){
	read_value[0] = array[pos_to_read];
}

void sparseMult(hipsparseHandle_t handle,
		hipsparseOperation_t transa,

		int m,
		int n,
		int k,
		int nnz,
		const float *alpha,
		const hipsparseMatDescr_t descrA,
		const float *csrValA,
		const int *csrRowPtrA,
		const int *csrColIndA,
		const float *B,
		int ldb,
		const float *beta,
		float *C,
		int ldc){


	float* csc_data;
	int* csc_ind;
	int* csc_ptr;

	hipblasStatus_t status = cublasAlloc(nnz, sizeof(float),
			(void**) &csc_data);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! device memory allocation error\n");
		exit(EXIT_FAILURE);
	}
	status = cublasAlloc(nnz, sizeof(int), (void**) &csc_ind);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! device memory allocation error\n");
		exit(EXIT_FAILURE);
	}
	status = cublasAlloc(k+1, sizeof(int), (void**) &csc_ptr);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! device memory allocation error\n");
		exit(EXIT_FAILURE);
	}

	hipsparseStatus_t cusparseStatus = hipsparseScsr2csc(handle, m,  k, nnz,
			csrValA, csrRowPtrA,
			csrColIndA, csc_data,
			csc_ind, csc_ptr,
			HIPSPARSE_ACTION_NUMERIC,
			HIPSPARSE_INDEX_BASE_ZERO);

	checkCudaErrors(cusparseStatus);

	cusparseStatus = hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE ,
			k, n, m , nnz,
			alpha,descrA,
			csc_data, csc_ptr, csc_ind,
			B,  ldb ,
			beta,
			C, ldc);
	checkCudaErrors(cusparseStatus);

	hipDeviceSynchronize();

	hipblasStatus_t status2 = cublasFree(csc_data);
	if (status2 != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! memory free error on delete _sparsePtr\n");
		exit(EXIT_FAILURE);
	}
	status2 = cublasFree(csc_ind);
	if (status2 != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! memory free error on delete _sparsePtr\n");
		exit(EXIT_FAILURE);
	}
	status2 = cublasFree(csc_ptr);
	if (status2 != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! memory free error on delete _sparsePtr\n");
		exit(EXIT_FAILURE);
	}

}

